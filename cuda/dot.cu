
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N (1024*1024)
#define M 1024

__global__ void dot(float *a, float *b, float *c) {
	int i = blockDim.x*blockIdx.x + threadIdx.x, j = threadIdx.x;
	__shared__ float ab[M];
	ab[j] = a[i]*b[i];
	__syncthreads();
	if (!j) {	
		float s = 0.;
		for (i = 0; i < M; i++)
			s += ab[i];
		atomicAdd(c, s);
	}
}

int main(int argc, char *argv[]) {
	int i = 0, size = N*sizeof(float);
	float *a, *b, *c, *dev_a, *dev_b, *dev_c;
	a = (float *) malloc(size);
	b = (float *) malloc(size);
	c = (float *) malloc(sizeof(float));
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_b, size);
	hipMalloc((void **) &dev_c, sizeof(float));

	for (i = 0; i < N; i++) {a[i] = 1.; b[i] = 1.;} *c = 0.;
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(float), hipMemcpyHostToDevice);
	dot<<<N/M, M>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, sizeof(float), hipMemcpyDeviceToHost);
	printf("%f\n", *c);

	hipFree(dev_c); hipFree(dev_b); hipFree(dev_a);
	free(c); free(b); free(a);
	return 0;
}
