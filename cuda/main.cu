
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#define abs(x) ( (x > 0) ? (x) : (-x) )
#define M 32*64
#define N 16*64
#define L 128*128
#define l 128

float *matrix_init(int m, int n);
void matrix_print(float *A, int m, int n);
void matrix_mult(float *A, int m, int q, float *B, int p, int n, float *C);

__global__ void dot(float *A, float *B, float *C) {
	int i, j;
	int I = blockDim.x*blockIdx.x + threadIdx.x, J = threadIdx.x, k;
	__shared__ float AB[l];
	float s = 0.;
	for (i = 0; i < M; i++)
		for (j = 0; j < N; j++) {
			AB[J] = A[i*L+I]*B[I*N+j];
			__syncthreads();
			if (!J) {
				s = 0.;
				for (k = 0; k < l; k++)
					s += AB[k];
				atomicAdd((C+i*N+j), s);
			}
		}
}

int main(int argc, char *argv[]) {
	int i, j, k = 0;	time_t dt, ht;
	float *A, *B, *C, *D, *_A, *_B, *_C;
	A = matrix_init(M, L);
	B = matrix_init(L, N);
	C = matrix_init(M, N);
	D = matrix_init(M, N);
	hipMalloc((void **) &_A, M*L*sizeof(float));
	hipMalloc((void **) &_B, L*N*sizeof(float));
	hipMalloc((void **) &_C, M*N*sizeof(float));
	srand(time(NULL));
	for (i = 0; i < M; i++)
		for (j = 0; j < L; j++) 
			*(A+i*L+j) = 1.;
	for (i = 0; i < L; i++)
		for (j = 0; j < N; j++) 
			*(B+i*N+j) = 1.;

	dt = time(NULL);
	hipMemcpy(_A, A, M*L*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(_B, B, L*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(_C, C, M*N*sizeof(float), hipMemcpyHostToDevice);
	dot<<<L/l, l>>>(_A, _B, _C);
	hipMemcpy(C, _C, M*N*sizeof(float), hipMemcpyDeviceToHost);
	dt = time(NULL) - dt;
	printf("device: %d sec\n", (int) dt);
	fflush(stdout);
	
	ht = time(NULL); 
	matrix_mult(A, M, L, B, L, N, D);
	ht = time(NULL) - ht; 
	printf("host: %d sec\n", (int) ht );		
	printf("acceleration: %.0lf\n", (((double)ht)/((double)dt)) );		

	for (i = 0; i < M; i++)
		for (j = 0; j < N; j++) 
			if(abs(*(C+i*N+j) - *(D+i*N+j)) > .0000001) 
				k++;
	printf("error: %d\n", k);

	hipFree(_C); hipFree(_B); hipFree(_A);
	free(C); free(B); free(A);
	return 0;
}

// Matrix functions
float *matrix_init(int m, int n) {
	int i, j;
	float *A = (float *) malloc (m*n*sizeof(float *));
	for (i = 0; i < m; i++)
		for (j = 0; j < n; j++)
			*(A+i*n+j) = 0.;
	return A;
}
void matrix_print(float *A, int m, int n) {
	if (!A) {printf("Empty!\n"); return;}
	int i = 0, j = 0;
	printf("\n");
	for  (i = 0 ; i < m; i++) {
		printf(" |");
		for (j = 0; j < n; j++) 
			printf( "%7lg" , *(A+i*n+j) );
		printf("%6c|",' ');
		printf("\n"); 
	}	printf("\n");
}
void matrix_mult(float *A, int m, int q, float *B, int p, int n, float *C) {
	if (q != p) {C = NULL; return;}
	int i = 0, j = 0, r = 0; 
	float s = 0.0;
	for (i = 0; i < m; i++) 
		for (j = 0; j < n; j++) {
			s = 0.0;
			for (r = 0; r < q; r++) 
				s += (*(A+i*q+r))*(*(B+r*n+j));
			*(C+i*n+j) = s;
		}
}	
